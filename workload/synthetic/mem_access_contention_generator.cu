#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <sys/syscall.h>
#include <unistd.h>
#include <sys/types.h>
#include <nvToolsExt.h>
#include <string.h>
#include <assert.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define MEMSIZE (0x1 << 22) //8*2M*4B=128MB

	__global__
void vertorADDGPU(float *x, float *y, float *z) {
	int idx = blockIdx.x * MEMSIZE + 32 * threadIdx.x;
	while(true){
		z[idx] = x[idx] + y[idx];
		z[idx + blockDim.x * 32] = x[idx + blockDim.x * 32] + y[idx + blockDim.x * 32];
		z[idx + 2 * blockDim.x * 32] = x[idx + 2 * blockDim.x * 32] + y[idx + 2 * blockDim.x * 32];
		z[idx + 3 * blockDim.x * 32] = x[idx + 3 * blockDim.x * 32] + y[idx + 3 * blockDim.x * 32];
		z[idx + 4 * blockDim.x * 32] = x[idx + 4 * blockDim.x * 32] + y[idx + 4 * blockDim.x * 32];
		z[idx + 5 * blockDim.x * 32] = x[idx + 5 * blockDim.x * 32] + y[idx + 5 * blockDim.x * 32];
		z[idx + 6 * blockDim.x * 32] = x[idx + 6 * blockDim.x * 32] + y[idx + 6 * blockDim.x * 32];
		z[idx + 7 * blockDim.x * 32] = x[idx + 7 * blockDim.x * 32] + y[idx + 7 * blockDim.x * 32];
		z[idx + 8 * blockDim.x * 32] = x[idx + 8 * blockDim.x * 32] + y[idx + 8 * blockDim.x * 32];
                z[idx + 9 * blockDim.x * 32] = x[idx + 9 * blockDim.x * 32] + y[idx + 9 * blockDim.x * 32];
                z[idx + 10 * blockDim.x * 32] = x[idx + 10 * blockDim.x * 32] + y[idx + 10 * blockDim.x * 32];
                z[idx + 11 * blockDim.x * 32] = x[idx + 11 * blockDim.x * 32] + y[idx + 11 * blockDim.x * 32];
                z[idx + 12 * blockDim.x * 32] = x[idx + 12 * blockDim.x * 32] + y[idx + 12 * blockDim.x * 32];
                z[idx + 13 * blockDim.x * 32] = x[idx + 13 * blockDim.x * 32] + y[idx + 13 * blockDim.x * 32];
		z[idx + 14 * blockDim.x * 32] = x[idx + 14 * blockDim.x * 32] + y[idx + 14 * blockDim.x * 32];
		z[idx + 15 * blockDim.x * 32] = x[idx + 15 * blockDim.x * 32] + y[idx + 15 * blockDim.x * 32];
		z[idx + 16 * blockDim.x * 32] = x[idx + 16 * blockDim.x * 32] + y[idx + 16 * blockDim.x * 32];
                z[idx + 17 * blockDim.x * 32] = x[idx + 17 * blockDim.x * 32] + y[idx + 17 * blockDim.x * 32];
                z[idx + 18 * blockDim.x * 32] = x[idx + 18 * blockDim.x * 32] + y[idx + 18 * blockDim.x * 32];
                z[idx + 19 * blockDim.x * 32] = x[idx + 19 * blockDim.x * 32] + y[idx + 19 * blockDim.x * 32];
                z[idx + 20 * blockDim.x * 32] = x[idx + 20 * blockDim.x * 32] + y[idx + 20 * blockDim.x * 32];
                z[idx + 21 * blockDim.x * 32] = x[idx + 21 * blockDim.x * 32] + y[idx + 21 * blockDim.x * 32];
                z[idx + 22 * blockDim.x * 32] = x[idx + 22 * blockDim.x * 32] + y[idx + 22 * blockDim.x * 32];
                z[idx + 23 * blockDim.x * 32] = x[idx + 23 * blockDim.x * 32] + y[idx + 23 * blockDim.x * 32];
                z[idx + 24 * blockDim.x * 32] = x[idx + 24 * blockDim.x * 32] + y[idx + 24 * blockDim.x * 32];
                z[idx + 25 * blockDim.x * 32] = x[idx + 25 * blockDim.x * 32] + y[idx + 25 * blockDim.x * 32];
                z[idx + 26 * blockDim.x * 32] = x[idx + 26 * blockDim.x * 32] + y[idx + 26 * blockDim.x * 32];
                z[idx + 27 * blockDim.x * 32] = x[idx + 27 * blockDim.x * 32] + y[idx + 27 * blockDim.x * 32];
                z[idx + 28 * blockDim.x * 32] = x[idx + 28 * blockDim.x * 32] + y[idx + 28 * blockDim.x * 32];
                z[idx + 29 * blockDim.x * 32] = x[idx + 29 * blockDim.x * 32] + y[idx + 29 * blockDim.x * 32];
		z[idx + 30 * blockDim.x * 32] = x[idx + 30 * blockDim.x * 32] + y[idx + 30 * blockDim.x * 32];
                z[idx + 31 * blockDim.x * 32] = x[idx + 31 * blockDim.x * 32] + y[idx + 31 * blockDim.x * 32];
		z[idx + 32 * blockDim.x * 32] = x[idx + 32 * blockDim.x * 32] + y[idx + 32 * blockDim.x * 32];
                z[idx + 33 * blockDim.x * 32] = x[idx + 33 * blockDim.x * 32] + y[idx + 33 * blockDim.x * 32];
                z[idx + 34 * blockDim.x * 32] = x[idx + 34 * blockDim.x * 32] + y[idx + 34 * blockDim.x * 32];
                z[idx + 35 * blockDim.x * 32] = x[idx + 35 * blockDim.x * 32] + y[idx + 35 * blockDim.x * 32];
                z[idx + 36 * blockDim.x * 32] = x[idx + 36 * blockDim.x * 32] + y[idx + 36 * blockDim.x * 32];
                z[idx + 37 * blockDim.x * 32] = x[idx + 37 * blockDim.x * 32] + y[idx + 37 * blockDim.x * 32];
                z[idx + 38 * blockDim.x * 32] = x[idx + 38 * blockDim.x * 32] + y[idx + 38 * blockDim.x * 32];
                z[idx + 39 * blockDim.x * 32] = x[idx + 39 * blockDim.x * 32] + y[idx + 39 * blockDim.x * 32];
                z[idx + 40 * blockDim.x * 32] = x[idx + 40 * blockDim.x * 32] + y[idx + 40 * blockDim.x * 32];
                z[idx + 41 * blockDim.x * 32] = x[idx + 41 * blockDim.x * 32] + y[idx + 41 * blockDim.x * 32];
                z[idx + 42 * blockDim.x * 32] = x[idx + 42 * blockDim.x * 32] + y[idx + 42 * blockDim.x * 32];
                z[idx + 43 * blockDim.x * 32] = x[idx + 43 * blockDim.x * 32] + y[idx + 43 * blockDim.x * 32];
                z[idx + 44 * blockDim.x * 32] = x[idx + 44 * blockDim.x * 32] + y[idx + 44 * blockDim.x * 32];
                z[idx + 45 * blockDim.x * 32] = x[idx + 45 * blockDim.x * 32] + y[idx + 45 * blockDim.x * 32];
                z[idx + 46 * blockDim.x * 32] = x[idx + 46 * blockDim.x * 32] + y[idx + 46 * blockDim.x * 32];
                z[idx + 47 * blockDim.x * 32] = x[idx + 47 * blockDim.x * 32] + y[idx + 47 * blockDim.x * 32];
                z[idx + 48 * blockDim.x * 32] = x[idx + 48 * blockDim.x * 32] + y[idx + 48 * blockDim.x * 32];
                z[idx + 49 * blockDim.x * 32] = x[idx + 49 * blockDim.x * 32] + y[idx + 49 * blockDim.x * 32];
                z[idx + 50 * blockDim.x * 32] = x[idx + 50 * blockDim.x * 32] + y[idx + 50 * blockDim.x * 32];
                z[idx + 51 * blockDim.x * 32] = x[idx + 51 * blockDim.x * 32] + y[idx + 51 * blockDim.x * 32];
                z[idx + 52 * blockDim.x * 32] = x[idx + 52 * blockDim.x * 32] + y[idx + 52 * blockDim.x * 32];
                z[idx + 53 * blockDim.x * 32] = x[idx + 53 * blockDim.x * 32] + y[idx + 53 * blockDim.x * 32];
                z[idx + 54 * blockDim.x * 32] = x[idx + 54 * blockDim.x * 32] + y[idx + 54 * blockDim.x * 32];
                z[idx + 55 * blockDim.x * 32] = x[idx + 55 * blockDim.x * 32] + y[idx + 55 * blockDim.x * 32];
                z[idx + 56 * blockDim.x * 32] = x[idx + 56 * blockDim.x * 32] + y[idx + 56 * blockDim.x * 32];
                z[idx + 57 * blockDim.x * 32] = x[idx + 57 * blockDim.x * 32] + y[idx + 57 * blockDim.x * 32];
                z[idx + 58 * blockDim.x * 32] = x[idx + 58 * blockDim.x * 32] + y[idx + 58 * blockDim.x * 32];
                z[idx + 59 * blockDim.x * 32] = x[idx + 59 * blockDim.x * 32] + y[idx + 59 * blockDim.x * 32];
                z[idx + 60 * blockDim.x * 32] = x[idx + 60 * blockDim.x * 32] + y[idx + 60 * blockDim.x * 32];
                z[idx + 61 * blockDim.x * 32] = x[idx + 61 * blockDim.x * 32] + y[idx + 61 * blockDim.x * 32];
		z[idx + 62 * blockDim.x * 32] = x[idx + 62 * blockDim.x * 32] + y[idx + 62 * blockDim.x * 32];
                z[idx + 63 * blockDim.x * 32] = x[idx + 63 * blockDim.x * 32] + y[idx + 63 * blockDim.x * 32];
		z[idx + 64 * blockDim.x * 32] = x[idx + 64 * blockDim.x * 32] + y[idx + 64 * blockDim.x * 32];
		z[idx + 65 * blockDim.x * 32] = x[idx + 65 * blockDim.x * 32] + y[idx + 65 * blockDim.x * 32];
		z[idx + 66 * blockDim.x * 32] = x[idx + 66 * blockDim.x * 32] + y[idx + 66 * blockDim.x * 32];
		z[idx + 67 * blockDim.x * 32] = x[idx + 67 * blockDim.x * 32] + y[idx + 67 * blockDim.x * 32];
		z[idx + 68 * blockDim.x * 32] = x[idx + 68 * blockDim.x * 32] + y[idx + 68 * blockDim.x * 32];
                z[idx + 69 * blockDim.x * 32] = x[idx + 69 * blockDim.x * 32] + y[idx + 69 * blockDim.x * 32];
                z[idx + 70 * blockDim.x * 32] = x[idx + 70 * blockDim.x * 32] + y[idx + 70 * blockDim.x * 32];
                z[idx + 71 * blockDim.x * 32] = x[idx + 71 * blockDim.x * 32] + y[idx + 71 * blockDim.x * 32];
                z[idx + 72 * blockDim.x * 32] = x[idx + 72 * blockDim.x * 32] + y[idx + 72 * blockDim.x * 32];
                z[idx + 73 * blockDim.x * 32] = x[idx + 73 * blockDim.x * 32] + y[idx + 73 * blockDim.x * 32];
		z[idx + 74 * blockDim.x * 32] = x[idx + 74 * blockDim.x * 32] + y[idx + 74 * blockDim.x * 32];
		z[idx + 75 * blockDim.x * 32] = x[idx + 75 * blockDim.x * 32] + y[idx + 75 * blockDim.x * 32];
		z[idx + 76 * blockDim.x * 32] = x[idx + 76 * blockDim.x * 32] + y[idx + 76 * blockDim.x * 32];
                z[idx + 77 * blockDim.x * 32] = x[idx + 77 * blockDim.x * 32] + y[idx + 77 * blockDim.x * 32];
                z[idx + 78 * blockDim.x * 32] = x[idx + 78 * blockDim.x * 32] + y[idx + 78 * blockDim.x * 32];
                z[idx + 79 * blockDim.x * 32] = x[idx + 79 * blockDim.x * 32] + y[idx + 79 * blockDim.x * 32];
                z[idx + 80 * blockDim.x * 32] = x[idx + 80 * blockDim.x * 32] + y[idx + 80 * blockDim.x * 32];
                z[idx + 81 * blockDim.x * 32] = x[idx + 81 * blockDim.x * 32] + y[idx + 81 * blockDim.x * 32];
                z[idx + 82 * blockDim.x * 32] = x[idx + 82 * blockDim.x * 32] + y[idx + 82 * blockDim.x * 32];
                z[idx + 83 * blockDim.x * 32] = x[idx + 83 * blockDim.x * 32] + y[idx + 83 * blockDim.x * 32];
                z[idx + 84 * blockDim.x * 32] = x[idx + 84 * blockDim.x * 32] + y[idx + 84 * blockDim.x * 32];
                z[idx + 85 * blockDim.x * 32] = x[idx + 85 * blockDim.x * 32] + y[idx + 85 * blockDim.x * 32];
                z[idx + 86 * blockDim.x * 32] = x[idx + 86 * blockDim.x * 32] + y[idx + 86 * blockDim.x * 32];
                z[idx + 87 * blockDim.x * 32] = x[idx + 87 * blockDim.x * 32] + y[idx + 87 * blockDim.x * 32];
                z[idx + 88 * blockDim.x * 32] = x[idx + 88 * blockDim.x * 32] + y[idx + 88 * blockDim.x * 32];
                z[idx + 89 * blockDim.x * 32] = x[idx + 89 * blockDim.x * 32] + y[idx + 89 * blockDim.x * 32];
		z[idx + 90 * blockDim.x * 32] = x[idx + 90 * blockDim.x * 32] + y[idx + 90 * blockDim.x * 32];
                z[idx + 91 * blockDim.x * 32] = x[idx + 91 * blockDim.x * 32] + y[idx + 91 * blockDim.x * 32];
		z[idx + 92 * blockDim.x * 32] = x[idx + 92 * blockDim.x * 32] + y[idx + 92 * blockDim.x * 32];
                z[idx + 93 * blockDim.x * 32] = x[idx + 93 * blockDim.x * 32] + y[idx + 93 * blockDim.x * 32];
                z[idx + 94 * blockDim.x * 32] = x[idx + 94 * blockDim.x * 32] + y[idx + 94 * blockDim.x * 32];
                z[idx + 95 * blockDim.x * 32] = x[idx + 95 * blockDim.x * 32] + y[idx + 95 * blockDim.x * 32];
                z[idx + 96 * blockDim.x * 32] = x[idx + 96 * blockDim.x * 32] + y[idx + 96 * blockDim.x * 32];
                z[idx + 97 * blockDim.x * 32] = x[idx + 97 * blockDim.x * 32] + y[idx + 97 * blockDim.x * 32];
                z[idx + 98 * blockDim.x * 32] = x[idx + 98 * blockDim.x * 32] + y[idx + 98 * blockDim.x * 32];
                z[idx + 99 * blockDim.x * 32] = x[idx + 99 * blockDim.x * 32] + y[idx + 99 * blockDim.x * 32];
                z[idx + 100 * blockDim.x * 32] = x[idx + 100 * blockDim.x * 32] + y[idx + 100 * blockDim.x * 32];
                z[idx + 101 * blockDim.x * 32] = x[idx + 101 * blockDim.x * 32] + y[idx + 101 * blockDim.x * 32];
                z[idx + 102 * blockDim.x * 32] = x[idx + 102 * blockDim.x * 32] + y[idx + 102 * blockDim.x * 32];
                z[idx + 103 * blockDim.x * 32] = x[idx + 103 * blockDim.x * 32] + y[idx + 103 * blockDim.x * 32];
                z[idx + 104 * blockDim.x * 32] = x[idx + 104 * blockDim.x * 32] + y[idx + 104 * blockDim.x * 32];
                z[idx + 105 * blockDim.x * 32] = x[idx + 105 * blockDim.x * 32] + y[idx + 105 * blockDim.x * 32];
                z[idx + 106 * blockDim.x * 32] = x[idx + 106 * blockDim.x * 32] + y[idx + 106 * blockDim.x * 32];
                z[idx + 107 * blockDim.x * 32] = x[idx + 107 * blockDim.x * 32] + y[idx + 107 * blockDim.x * 32];
                z[idx + 108 * blockDim.x * 32] = x[idx + 108 * blockDim.x * 32] + y[idx + 108 * blockDim.x * 32];
                z[idx + 109 * blockDim.x * 32] = x[idx + 109 * blockDim.x * 32] + y[idx + 109 * blockDim.x * 32];
                z[idx + 110 * blockDim.x * 32] = x[idx + 110 * blockDim.x * 32] + y[idx + 110 * blockDim.x * 32];
                z[idx + 111 * blockDim.x * 32] = x[idx + 111 * blockDim.x * 32] + y[idx + 111 * blockDim.x * 32];
                z[idx + 112 * blockDim.x * 32] = x[idx + 112 * blockDim.x * 32] + y[idx + 112 * blockDim.x * 32];
                z[idx + 113 * blockDim.x * 32] = x[idx + 113 * blockDim.x * 32] + y[idx + 113 * blockDim.x * 32];
                z[idx + 114 * blockDim.x * 32] = x[idx + 114 * blockDim.x * 32] + y[idx + 114 * blockDim.x * 32];
                z[idx + 115 * blockDim.x * 32] = x[idx + 115 * blockDim.x * 32] + y[idx + 115 * blockDim.x * 32];
                z[idx + 116 * blockDim.x * 32] = x[idx + 116 * blockDim.x * 32] + y[idx + 116 * blockDim.x * 32];
                z[idx + 117 * blockDim.x * 32] = x[idx + 117 * blockDim.x * 32] + y[idx + 117 * blockDim.x * 32];
                z[idx + 118 * blockDim.x * 32] = x[idx + 118 * blockDim.x * 32] + y[idx + 118 * blockDim.x * 32];
                z[idx + 119 * blockDim.x * 32] = x[idx + 119 * blockDim.x * 32] + y[idx + 119 * blockDim.x * 32];
                z[idx + 120 * blockDim.x * 32] = x[idx + 120 * blockDim.x * 32] + y[idx + 120 * blockDim.x * 32];
                z[idx + 121 * blockDim.x * 32] = x[idx + 121 * blockDim.x * 32] + y[idx + 121 * blockDim.x * 32];
		z[idx + 122 * blockDim.x * 32] = x[idx + 122 * blockDim.x * 32] + y[idx + 122 * blockDim.x * 32];
                z[idx + 123 * blockDim.x * 32] = x[idx + 123 * blockDim.x * 32] + y[idx + 123 * blockDim.x * 32];
                z[idx + 124 * blockDim.x * 32] = x[idx + 124 * blockDim.x * 32] + y[idx + 124 * blockDim.x * 32];
                z[idx + 125 * blockDim.x * 32] = x[idx + 125 * blockDim.x * 32] + y[idx + 125 * blockDim.x * 32];
                z[idx + 126 * blockDim.x * 32] = x[idx + 126 * blockDim.x * 32] + y[idx + 126 * blockDim.x * 32];
                z[idx + 127 * blockDim.x * 32] = x[idx + 127 * blockDim.x * 32] + y[idx + 127 * blockDim.x * 32];
	}
}

int main(int argc, char *argv[]) {
	int thread_num;
	float *x, *y, *z, *d_x, *d_y, *d_z;
	unsigned int flags = hipHostMallocMapped;

	// Check the number of arguments
	if (argc != 2) {
		printf("please use with one argument\n");
		printf("The argument is thread num\n");
		return 1;
	}
	thread_num = atoi(argv[1]);

	// Dynamic memory allocation by using zero-copy memory (cache is off)
	hipHostAlloc((void **)&x, 8 * MEMSIZE * sizeof(float), flags);
	hipHostAlloc((void **)&y, 8 * MEMSIZE * sizeof(float), flags);
	hipHostAlloc((void **)&z, 8 * MEMSIZE * sizeof(float), flags);

	// Initialize the two input arrays
	for (int i = 0; i < 8 * MEMSIZE; i++) {
        	x[i] = rand() / (float) RAND_MAX;
        	y[i] = rand() / (float) RAND_MAX;
	}

	hipHostGetDevicePointer((void **)&d_x, (void *)x, 0);
	hipHostGetDevicePointer((void **)&d_y, (void *)y, 0);
	hipHostGetDevicePointer((void **)&d_z, (void *)z, 0);
	
	// Set the number of threads
	dim3 numBlocks(8);
	dim3 threadsPerBlock(thread_num);

	while(true){
		vertorADDGPU<<<numBlocks, threadsPerBlock>>>(d_x, d_y, d_z);
		hipDeviceSynchronize();
	}

	hipFree(d_x);
	hipFree(d_y);
	hipFree(d_z);
	return 1;
}
